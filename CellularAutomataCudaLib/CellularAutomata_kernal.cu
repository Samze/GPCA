
#include <hip/hip_runtime.h>
//#include "cellularautomata_kernal_DLL.h"

template <typename CAFunction>
__global__ void kernal(unsigned int* g_data, int* DIM, CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( !(x > *DIM) &&  !(y > *DIM)) {//Guard against launching too many threads
	//set new cell state.
	
		//__syncthreads();

		g_data[(x * *DIM) + y] = func->applyFunction(g_data,x,y,*DIM);
	
		//g_data[(x * *DIM) + y] = (x * *DIM) + y;
	}

}