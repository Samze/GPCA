
#include <hip/hip_runtime.h>
/*	GPCA - A Cellular Automata library powered by CUDA. 
    Copyright (C) 2011  Sam Gunaratne University of Plymouth

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

template <typename CAFunction>
__global__ void kernal(unsigned int* g_data, int* DIM, CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( !(x > *DIM) &&  !(y > *DIM)) {//Guard against launching too many threads
	//set new cell state.
	
		//__syncthreads();

		g_data[(x * *DIM) + y] = func->applyFunction(g_data,x,y,*DIM);
	
		//g_data[(x * *DIM) + y] = (x * *DIM) + y;
	}
}

template <typename CAFunction>
__global__ void kernal3DTest(unsigned int* g_data, int* DIM, CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	
	int slice = *DIM/blockDim.y + 1;

	int y = (blockIdx.y % slice) * blockDim.y + threadIdx.y;
	int z = blockIdx.y/slice;


	if( x >= *DIM ||  y >= *DIM || z >= *DIM) //Guard against launching too many threads
		return;
	
	//__syncthreads();

	g_data[(z * *DIM * *DIM) + (x * *DIM) + y] = func->applyFunction(func->pFlatGrid,x,y,z,*DIM);
	
}

template <typename CAFunction>
__global__ void kernal3D(unsigned int* g_data, int* DIM, CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z;// + blockIdx.z * blockDim.z;

	//This is our fake z area
	if( blockIdx.x >= gridDim.x/2) {
		//z = threadIdx.z + blockDim.z;
		x = threadIdx.x + (blockIdx.x - gridDim.x/2) * blockDim.x;
	}

	if( blockIdx.y >= gridDim.y/2) {
		//z = threadIdx.z + 2;
		y = threadIdx.y + (blockIdx.y - gridDim.y/2) * blockDim.y;
	}


	if( !(x > *DIM) &&  !(y > *DIM) && !(z > *DIM)) {//Guard against launching too many threads
	//set new cell state.
	
		//__syncthreads();

		g_data[(z * *DIM * *DIM) + (x * *DIM) + y] = func->applyFunction(g_data,x,y,z,*DIM);
		
	/*	g_data[0] = 1;*/
		//g_data[(x * *DIM) + y] = (x * *DIM) + y;
	}
}
