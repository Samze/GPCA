
#include <hip/hip_runtime.h>
/*	GPCA - A Cellular Automata library powered by CUDA. 
    Copyright (C) 2011  Sam Gunaratne University of Plymouth

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
template <typename CAFunction>
__global__ void kernal(CAFunction* func) {
	
	//global x/y positions
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;


	int DIM = func->lattice->DIM;
	void* grid = func->lattice->pFlatGrid;
	

	if( !(x > DIM) &&  !(y > DIM)) {//Guard against launching too many threads
			
		func->applyFunction(grid,x,y,DIM);
		//grid[x * DIM + y] = result;
	}
}

template <typename CAFunction>
__global__ void kernalSharedMem(CAFunction* func) {
	
	//global x/y positions
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;

	//block x/y positions (adjusting for launching more threads than needed for the update of states)
	int bx = threadIdx.x - 1;
	int by = threadIdx.y - 1;

	int DIM = func->lattice->DIM;

	//22 needs to mirror how many threads launched in the kernal...can't use blockDim.x/y.
	__shared__ unsigned int shar_data[22 * 22];

	unsigned int* grid = (unsigned int*)func->lattice->pFlatGrid;
	
	//2 because of the padding!
	int xOrigin = (x - 1) - (blockIdx.x * 2);
	int yOrigin = (y - 1) - (blockIdx.y * 2);

	//checking bounds..
	if(x != 0 && y != 0 && x != blockDim.x * gridDim.x && y != blockDim.y * gridDim.y) {

		shar_data[threadIdx.x * blockDim.x + threadIdx.y] = grid[xOrigin * DIM + yOrigin];

	} else {
		shar_data[threadIdx.x * blockDim.x + threadIdx.y] = 0;
	}

	//void* grid = func->lattice->pFlatGrid;
	__syncthreads();

	
	//We only want to update the state of cells in our 'inner area'
	if(bx >= 0 && bx < 20 && by >= 0 && by < 20) {

		func->applyFunction(shar_data,threadIdx.x,threadIdx.y,blockDim.x);
		
		//__syncthreads();
		grid[xOrigin * DIM + yOrigin] = shar_data[threadIdx.x * blockDim.x + threadIdx.y];

	}
		

}

template <typename CAFunction>
__global__ void SCIARAKernal(CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	int DIM = func->lattice->DIM;
	void* grid = func->lattice->pFlatGrid;

	if( !(x > DIM) &&  !(y > DIM)) {//Guard against launching too many threads
	//set new cell state.
	
		//__syncthreads();

		//grid[(x * DIM) + y] = func->applyFunction(grid,x,y,DIM);
	
		//__syncthreads();
		
		//grid[(x * DIM) + y] = func->computethickness(grid,x,y,DIM);

		//grid[(x * DIM) + y] = (976562499 << func->lattice->noBits);
		//g_data[(x * *DIM) + y] = (x * *DIM) + y;
	}
}

template <typename CAFunction>
__global__ void SCIARAKernal2(CAFunction* func) {
	
	int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	int DIM = func->lattice->DIM;
	void* grid = func->lattice->pFlatGrid;

	if( !(x > DIM) &&  !(y > DIM)) {//Guard against launching too many threads
	//set new cell state.
	
		//__syncthreads();

		func->applyFunction(grid,x,y,DIM);
	
		//__syncthreads();
		
		func->computethickness(grid,x,y,DIM);

		//grid[(x * DIM) + y] = (976562499 << func->lattice->noBits);
		//g_data[(x * *DIM) + y] = (x * *DIM) + y;
	}
}

template <typename CAFunction>
__global__ void kernal3D(CAFunction* func) {
	int DIM = func->lattice->DIM;
	void* grid = (unsigned int*)func->lattice->pFlatGrid;

	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	
	int slice = DIM/blockDim.y + 1;

	int y = (blockIdx.y % slice) * blockDim.y + threadIdx.y;
	int z = blockIdx.y/slice;

	//TODO fix coding style inconsistancy.
	if( x >= DIM ||  y >= DIM || z >= DIM) //Guard against launching too many threads
		return;
	

//	grid[(z * DIM * DIM) + (x * DIM) + y] = func->applyFunction(grid,x,y,z,DIM);
	func->applyFunction(grid,x,y,z,DIM);
	
}

template <typename CAFunction>
__global__ void kernal3DTest(CAFunction* func) {
	int DIM = func->lattice->DIM;
	unsigned int* grid = (unsigned int*)func->lattice->pFlatGrid;

	int blockSlice = blockIdx.x / gridDim.y;


	int x = threadIdx.x + (blockIdx.x - (blockSlice * gridDim.y))  * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y; 
	int z = threadIdx.z + blockSlice * blockDim.x;



	//TODO fix coding style inconsistancy.
	if( x >= DIM ||  y >= DIM || z >= DIM) //Guard against launching too many threads
		return;


	func->applyFunction(grid,x,y,z,DIM);
}


template <typename CAFunction>
__global__ void kernal3DTestShared(CAFunction* func) {
	int DIM = func->lattice->DIM;
	unsigned int* grid = (unsigned int*)func->lattice->pFlatGrid;

	int blockSlice = blockIdx.x / gridDim.y;


	int x = threadIdx.x + (blockIdx.x - (blockSlice * gridDim.y))  * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y; 
	int z = threadIdx.z + blockSlice * blockDim.z;

	int bx = threadIdx.x - 1;
	int by = threadIdx.y - 1;
	int bz = threadIdx.z - 1;
	
	//8*8*8, 512 is the maximum size of a block..
	__shared__ unsigned int shar_data[512];

	//2 because of the padding!
	int xOrigin = (x - 1) - ((blockIdx.x - (blockSlice * gridDim.y)) * 2);
	int yOrigin = (y - 1) - (blockIdx.y * 2);
	int zOrigin = (z - 1) - (blockSlice * 2);



	int sharPos = (threadIdx.z * pow(blockDim.z,2.0f)) + (threadIdx.x * blockDim.x) + threadIdx.y;
	
	int globPos = (zOrigin * DIM * DIM ) + (xOrigin * DIM) + yOrigin;


	//checking bounds..
	if(x != 0 && y != 0 && x != blockDim.x * gridDim.y && y != blockDim.y * gridDim.y && z != 0 && z != blockDim.x * gridDim.y) {
		shar_data[sharPos] = grid[globPos];
	} 
	else{
		shar_data[sharPos] = 0;
	}

	//void* grid = func->lattice->pFlatGrid;
	//__syncthreads();

	//
	////We only want to update the state of cells in our 'inner area'
	//if(bx >= 0 && bx < 6 && by >= 0 && by < 6 && bz >= 0 && bz < 6) {

	//	//grid[globPos] = 3;

	//	func->applyFunction(shar_data,threadIdx.x,threadIdx.y,threadIdx.z,blockDim.y);

	//	grid[globPos] = shar_data[sharPos];

	//}


	//TODO fix coding style inconsistancy.
	//if( x >= DIM ||  y >= DIM || z >= DIM) //Guard against launching too many threads
	//	return;
	

//	grid[(z * DIM * DIM) + (x * DIM) + y] = func->applyFunction(grid,x,y,z,DIM);
	//func->applyFunction(grid,x,y,z,DIM);
}