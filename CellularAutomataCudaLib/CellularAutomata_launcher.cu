#include "hip/hip_runtime.h"
/*	GPCA - A Cellular Automata library powered by CUDA. 
    Copyright (C) 2011  Sam Gunaratne University of Plymouth

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "CellularAutomata_launcher.h"

#include "CellularAutomata_kernal.cu"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include "Abstract2DCA.h"
#include "Abstract3DCA.h"
#include <vector>

//TEMP REMOVE LATER
#include "SCIARA2.h"

template<typename CAFunction>
extern float CUDATimeStepSCIARA2(CAFunction *func) {

	void* *dev_pFlatGrid; //Pointers to device allocated memory

	CAFunction *dev_func;
	Abstract2DCA *dev_lattice;

	Abstract2DCA *tempLattice;
	void* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	int DIM = func->lattice->DIM;

	//TODO remove this..
	size_t noCells = DIM * DIM * sizeof(SCIARA2::Cell);

	//Might need to flatten the 2d array ormaybe try "int2" type

	//TODO fix this name
	size_t size = sizeof(CAFunction);
	size_t sizeLattice = sizeof(Abstract2DCA);//func->lattice2->size();
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);


	//Make our 2D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	dim3 threads(16,16);
	dim3 blocks (DIM/threads.x + 1,(DIM/threads.y + 1) * DIM);


	hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;

	func->lattice->pFlatGrid = dev_pFlatGrid;

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);

	//We want to temporarily hold our pointers so we can reassign them after the object copy...

	tempLattice = func->lattice;

	//reassign our pointers so we know where we put our dynamic arrays
	func->lattice = dev_lattice;

	
	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	SCIARAKernal2<<<blocks,threads>>>(dev_func);

	//Copy back to host
	hipMemcpy(tempGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//Reassign our dynamic array pointers
	func->lattice = tempLattice;
	func->lattice->pFlatGrid = tempGrid;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	//for (int i = 0; i < DIM; ++i) {
	//	for (int j = 0; j < DIM; ++j) {
	//			func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
	//	}
	//}

	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_func);

	return elapsedTime;
}


template<typename CAFunction>
extern float CUDATimeStepSCIARA(CAFunction *func) {

	void* dev_pFlatGrid; //Pointers to device allocated memory

	CAFunction *dev_func;
	Abstract2DCA *dev_lattice;

	Abstract2DCA *tempLattice;
	void* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	int DIM = func->lattice->DIM;

	size_t noCells = DIM * DIM * sizeof(unsigned int);

	//Might need to flatten the 2d array ormaybe try "int2" type

	//TODO fix this name
	size_t size = sizeof(CAFunction);
	size_t sizeLattice = sizeof(Abstract2DCA);//func->lattice2->size();
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);


	//Make our 2D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	dim3 blocks (DIM/20,DIM/20);
	dim3 threads(20,20);


	hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;

	func->lattice->pFlatGrid = dev_pFlatGrid;

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);

	//We want to temporarily hold our pointers so we can reassign them after the object copy...

	tempLattice = func->lattice;

	//reassign our pointers so we know where we put our dynamic arrays
	func->lattice = dev_lattice;

	
	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	SCIARAKernal<<<blocks,threads>>>(dev_func);

	//Copy back to host
	hipMemcpy(tempGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//Reassign our dynamic array pointers
	func->lattice = tempLattice;
	func->lattice->pFlatGrid = tempGrid;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	//for (int i = 0; i < DIM; ++i) {
	//	for (int j = 0; j < DIM; ++j) {
	//			func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
	//	}
	//}

	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_func);

	return elapsedTime;
}

template<typename CAFunction>
extern float CUDATimeStep(CAFunction *func) {

	void* dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_born; //to bornNo
	int *dev_survive; //to surviveNo
	CAFunction *dev_func;
	Abstract2DCA *dev_lattice;

	int* tempBorn;
	int* tempSurv;
	Abstract2DCA *tempLattice;
	void* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	int DIM = func->lattice->DIM;

	size_t noCells = DIM * DIM * sizeof(unsigned int);

	//Might need to flatten the 2d array ormaybe try "int2" type

	//TODO fix this name
	size_t size = sizeof(CAFunction);
	size_t sizeLattice = sizeof(Abstract2DCA);//func->lattice2->size();
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);

	hipMalloc((void**) &dev_born, sizeof(int) * func->bornSize);
	hipMalloc((void**) &dev_survive, sizeof(int) * func->surviveSize);

	//Make our 2D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	dim3 blocks (DIM/20,DIM/20);
	dim3 threads(20,20);


	//copy our two dynamic arrays 
	hipMemcpy(dev_born, func->bornNo, sizeof(int) * func->bornSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_survive, func->surviveNo, sizeof(int) * func->surviveSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;

	func->lattice->pFlatGrid = dev_pFlatGrid;

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);

	//We want to temporarily hold our pointers so we can reassign them after the object copy...
	tempBorn = func->bornNo;
	tempSurv = func->surviveNo;
	tempLattice = func->lattice;

	//reassign our pointers so we know where we put our dynamic arrays
	func->surviveNo = dev_survive;
	func->bornNo = dev_born;
	func->lattice = dev_lattice;

	
	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	kernal<<<blocks,threads>>>(dev_func);

	//Copy back to host
	hipMemcpy(tempGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//Reassign our dynamic array pointers
	func->surviveNo = tempSurv;
	func->bornNo = tempBorn;
	func->lattice = tempLattice;
	func->lattice->pFlatGrid = tempGrid;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	for (int i = 0; i < DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				//func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
		}
	}

	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_born);
	hipFree(dev_survive);
	hipFree(dev_func);

	return elapsedTime;
}

template<typename CAFunction>
extern float CUDATimeStep3D(CAFunction *func) {

	//unsigned int *dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_born; //to bornNo
	int *dev_survive; //to surviveNo
	unsigned int* dev_neighCount;
	CAFunction *dev_func;
	Abstract3DCA *dev_lattice;

	int* tempBorn;
	int* tempSurv;
	unsigned int* tempNeigh;
	Abstract3DCA *tempLattice;
	void* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	
	int DIM = func->lattice->DIM;

	size_t noCells = DIM * DIM * DIM * sizeof(unsigned int);
	//Might need to flatten the 2d array ormaybe try "int2" type
	
	//TODO fix this name
	size_t size = sizeof(CAFunction);

	map<void**, size_t>* hostDynamicMap = func->getDynamicArrays();

	map<void**, size_t>::const_iterator iter;
	
	map<void**,void*> tempPointers;

	for(iter = hostDynamicMap->begin(); iter != hostDynamicMap->end(); ++iter) {

		void** tempPointer = (*iter).first;
		void* dataPointer = *(*iter).first;
		tempPointers.insert(make_pair(tempPointer,dataPointer));
	}

	vector<void*>* devicePointers = setupDynamicArrays(*hostDynamicMap);

	//TODO Add this 
	//size_t sizeLattice = func->lattice->size();
	size_t sizeLattice = sizeof(Abstract3DCA);


	//Allocate suitable size memory on device
//	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);

	//hipMalloc((void**) &dev_born, sizeof(int) * func->bornSize);
	//hipMalloc((void**) &dev_survive, sizeof(int) * func->surviveSize);
	hipMalloc((void**) &dev_neighCount, noCells);


	//Do our specific setup, such as copying any dynamic arrays we may require.
	//func->setup();

	//Make our 3D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	/*dim3 blocks (1,1,1);
	dim3 threads(8,8,8);*/
	
	dim3 threads(16,16);
	dim3 blocks (DIM/threads.x + 1,(DIM/threads.y + 1) * DIM);

	//copy our two dynamic arrays 
	//hipMemcpy(dev_born, func->bornNo, sizeof(int) * func->bornSize,
	//	hipMemcpyHostToDevice);
	//hipMemcpy(dev_survive, func->surviveNo, sizeof(int) * func->surviveSize,
	//	hipMemcpyHostToDevice);
	//hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
	//	hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;
	tempNeigh = func->lattice->neighbourCount;
/*
	func->lattice->pFlatGrid = dev_pFlatGrid;*/
	func->lattice->neighbourCount = dev_neighCount;

	//reassign our pointers so we know where we put our dynamic arrays
	int count = 0;

	map<void**, void*>::const_iterator iterTP;
	
	for(iterTP = tempPointers.begin(); iterTP != tempPointers.end(); ++iterTP) {

		void** tmpPointer = (*iterTP).first;

		*tmpPointer = devicePointers->at(count);
		++count;
	}

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);
	
	tempLattice = func->lattice;
	func->lattice = dev_lattice;
	//func->surviveNo = dev_survive;
	//func->bornNo = dev_born;

	//We want to temporarily hold our pointers so we can reassign them after the object copy...
	tempBorn = func->bornNo;
	tempSurv = func->surviveNo;


	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	kernal3DTest<<<blocks,threads>>>(dev_func);



	//Because of our func currently holding a device pointer, we need to use a
	//temp pointer.
	hipMemcpy(tempNeigh, dev_neighCount, noCells,
		hipMemcpyDeviceToHost);


	//Reassign our dynamic pointers
	count = 0;

	for(iterTP = tempPointers.begin(); iterTP != tempPointers.end(); ++iterTP) {

		void** pointerLoc = (*iterTP).first;
		void* tmpPointer = (*iterTP).second;
		
		size_t size = hostDynamicMap->at(pointerLoc);

		hipMemcpy(tmpPointer, devicePointers->at(count), size,
			hipMemcpyDeviceToHost);

		*pointerLoc = tmpPointer;
		//Copy back to host


		//pointerLoc = &tmpPointer;

		hipFree(devicePointers->at(count));

		++count;
	}
	
	//func->surviveNo = tempSurv;
	//func->bornNo = tempBorn;

	func->lattice = tempLattice;
	//func->lattice->pFlatGrid = tempGrid;
	func->lattice->neighbourCount = tempNeigh;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize, this could be another kernal really..

	for (int i = 0; i < DIM * DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				//func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
		}
	}

	//Free memory on Device
//	hipFree(dev_pFlatGrid);
	//hipFree(dev_born);
	//hipFree(dev_survive);
	hipFree(dev_func);
	hipFree(dev_neighCount);

	return elapsedTime;
}
