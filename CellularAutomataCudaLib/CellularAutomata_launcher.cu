#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include "CellularAutomata_kernal.cu"


template<typename CAFunction>
extern float CUDATimeStep(int* pFlatGrid, int DIM, CAFunction *func) {

	int *dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_DIM;
	int *dev_born; //to bornNo
	int *dev_survive; //to surviveNo
	CAFunction *dev_func;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	size_t noCells = DIM * DIM * sizeof(int);
	//Might need to flatten the 2d array ormaybe try "int2" type
	
	//TODO fix this name
	size_t size = sizeof(CAFunction);
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_DIM, sizeof(int));
	hipMalloc((void**) &dev_func, sizeof(CAFunction));

	hipMalloc((void**) &dev_born, sizeof(int) * func->bornSize);
	hipMalloc((void**) &dev_survive, sizeof(int) * func->surviveSize);

	//Make our 2D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	dim3 blocks (DIM/20,DIM/20);
	dim3 threads(20,20);


	//copy our two dynamic arrays 
	hipMemcpy(dev_survive, func->surviveNo, sizeof(int) * func->surviveSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_born, func->bornNo, sizeof(int) * func->bornSize,
		hipMemcpyHostToDevice);

	//reassign our pointers so we know where we put our dynamic arrays
	func->surviveNo = dev_survive;
	func->bornNo = dev_born;
	
	
	
	//Copy our memory from Host to Device
	hipMemcpy(dev_pFlatGrid, pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_DIM, &DIM, sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_func, func, sizeof(CAFunction),
		hipMemcpyHostToDevice);


	kernal<<<blocks,threads>>>(dev_pFlatGrid, dev_DIM, dev_func);

	//Copy back to host
	hipMemcpy(pFlatGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	for (int i = 0; i < DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				pFlatGrid[i * DIM +j] = pFlatGrid[i * DIM +j] >> func->noBits;
		}
	}


	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_DIM);
	hipFree(dev_func);
	hipFree(dev_born);
	hipFree(dev_survive);

	return elapsedTime;
}

