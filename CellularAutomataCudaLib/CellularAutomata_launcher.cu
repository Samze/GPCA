#include "hip/hip_runtime.h"
/*	GPCA - A Cellular Automata library powered by CUDA. 
    Copyright (C) 2011  Sam Gunaratne University of Plymouth

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "CellularAutomata_kernal.cu"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include "Abstract2DCA.h"
#include "Abstract3DCA.h"

template<typename CAFunction>
extern float CUDATimeStep(CAFunction *func) {

	unsigned int *dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_born; //to bornNo
	int *dev_survive; //to surviveNo
	CAFunction *dev_func;
	Abstract2DCA *dev_lattice;

	int* tempBorn;
	int* tempSurv;
	Abstract2DCA *tempLattice;
	unsigned int* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	int DIM = func->lattice->DIM;

	size_t noCells = DIM * DIM * sizeof(unsigned int);

	//Might need to flatten the 2d array ormaybe try "int2" type
	
	//TODO fix this name
	size_t size = sizeof(CAFunction);
	size_t sizeLattice = sizeof(Abstract2DCA);//func->lattice2->size();
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);

	hipMalloc((void**) &dev_born, sizeof(int) * func->bornSize);
	hipMalloc((void**) &dev_survive, sizeof(int) * func->surviveSize);

	//Make our 2D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	dim3 blocks (DIM/20,DIM/20);
	dim3 threads(20,20);


	//copy our two dynamic arrays 
	hipMemcpy(dev_born, func->bornNo, sizeof(int) * func->bornSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_survive, func->surviveNo, sizeof(int) * func->surviveSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;

	func->lattice->pFlatGrid = dev_pFlatGrid;

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);

	//We want to temporarily hold our pointers so we can reassign them after the object copy...
	tempBorn = func->bornNo;
	tempSurv = func->surviveNo;
	tempLattice = func->lattice;

	//reassign our pointers so we know where we put our dynamic arrays
	func->surviveNo = dev_survive;
	func->bornNo = dev_born;
	func->lattice = dev_lattice;

	
	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	kernal<<<blocks,threads>>>(dev_func);

	//Copy back to host
	hipMemcpy(tempGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//Reassign our dynamic array pointers
	func->surviveNo = tempSurv;
	func->bornNo = tempBorn;
	func->lattice = tempLattice;
	func->lattice->pFlatGrid = tempGrid;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	for (int i = 0; i < DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
		}
	}

	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_born);
	hipFree(dev_survive);
	hipFree(dev_func);

	return elapsedTime;
}

template<typename CAFunction>
extern float CUDATimeStep3D(CAFunction *func) {

	unsigned int *dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_born; //to bornNo
	int *dev_survive; //to surviveNo
	unsigned int* dev_neighCount;
	CAFunction *dev_func;
	Abstract3DCA *dev_lattice;

	int* tempBorn;
	int* tempSurv;
	unsigned int* tempNeigh;
	Abstract3DCA *tempLattice;
	unsigned int* tempGrid;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	
	int DIM = func->lattice->DIM;

	size_t noCells = DIM * DIM * DIM * sizeof(unsigned int);
	//Might need to flatten the 2d array ormaybe try "int2" type
	
	//TODO fix this name
	size_t size = sizeof(CAFunction);

	//TODO Add this 
	//size_t sizeLattice = func->lattice->size();
	size_t sizeLattice = sizeof(Abstract3DCA);

	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_func, size);
	hipMalloc((void**) &dev_lattice, sizeLattice);

	hipMalloc((void**) &dev_born, sizeof(int) * func->bornSize);
	hipMalloc((void**) &dev_survive, sizeof(int) * func->surviveSize);
	hipMalloc((void**) &dev_neighCount, noCells);


	//Make our 3D grid of blocks & threads (DIM/No of threads)
	//One pixel is one thread.
	/*dim3 blocks (1,1,1);
	dim3 threads(8,8,8);*/
	
	dim3 threads(16,16);
	dim3 blocks (DIM/threads.x + 1,(DIM/threads.y + 1) * DIM);

	//copy our two dynamic arrays 
	hipMemcpy(dev_born, func->bornNo, sizeof(int) * func->bornSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_survive, func->surviveNo, sizeof(int) * func->surviveSize,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_pFlatGrid, func->lattice->pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	
	tempGrid = func->lattice->pFlatGrid;
	tempNeigh = func->lattice->neighbourCount;

	func->lattice->pFlatGrid = dev_pFlatGrid;
	func->lattice->neighbourCount = dev_neighCount;

	hipMemcpy(dev_lattice, func->lattice, sizeLattice,
		hipMemcpyHostToDevice);

	//We want to temporarily hold our pointers so we can reassign them after the object copy...
	tempBorn = func->bornNo;
	tempSurv = func->surviveNo;
	tempLattice = func->lattice;

	//reassign our pointers so we know where we put our dynamic arrays
	func->surviveNo = dev_survive;
	func->bornNo = dev_born;
	func->lattice = dev_lattice;
	
	//Copy our memory from Host to Device
	hipMemcpy(dev_func, func,size,
		hipMemcpyHostToDevice);

	kernal3DTest<<<blocks,threads>>>(dev_func);

	//Copy back to host
	hipMemcpy(tempGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//Because of our func currently holding a device pointer, we need to use a
	//temp pointer.
	hipMemcpy(tempNeigh, dev_neighCount, noCells,
		hipMemcpyDeviceToHost);


	//Reassign our dynamic array pointers
	func->surviveNo = tempSurv;
	func->bornNo = tempBorn;

	func->lattice = tempLattice;
	func->lattice->pFlatGrid = tempGrid;
	func->lattice->neighbourCount = tempNeigh;

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize, this could be another kernal really..

	for (int i = 0; i < DIM * DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				func->lattice->pFlatGrid[i * DIM +j] = func->lattice->pFlatGrid[i * DIM +j] >> func->lattice->getNoBits();
		}
	}


	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_born);
	hipFree(dev_survive);
	hipFree(dev_func);
	hipFree(dev_neighCount);

	return elapsedTime;
}

//TODO add support for this.
//const char* errorCheck() {
//	hipError_t err = hipGetLastError();
//	if (err != hipSuccess) {
//		return hipGetErrorString(err);
//	}
//	return  NULL;
//}
